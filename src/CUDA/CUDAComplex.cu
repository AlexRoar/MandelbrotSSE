
#include <hip/hip_runtime.h>
template<typename T>
struct hipComplex {
    T r;
    T i;

    hipComplex(T a, T b) : r(a), i(b) {}

    T magnitude2() {
        return r * r + i * i;
    }

    hipComplex operator*(const hipComplex &a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }

    hipComplex operator+(const hipComplex &a) {
        return hipComplex(r + a.r, i + a.i);
    }

    T abs() const {
        return sqrt(magnitude2());
    }
};